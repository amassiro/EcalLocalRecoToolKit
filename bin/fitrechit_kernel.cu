#include "hip/hip_runtime.h"
#include "fitrechit_kernel.cuh"


__global__ void 
kernel_print() {
  printf("Hello!\n");
}

void non_kernel_print(int nblocks, int nthreads_per_block) {
  kernel_print<<<nblocks, nthreads_per_block>>>();
}




