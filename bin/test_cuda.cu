#include <cstdio>
#include <cstdlib>
#include <string>
#include <hip/hip_runtime.h>

#include <iostream>


__global__
void print(const char * message, size_t length)
{
  //printf("blockIdx.x, threadIdx.x: %d, %d\n", blockIdx.x, threadIdx.x);
  for (size_t i = blockDim.x * blockIdx.x + threadIdx.x; i < length; i += blockDim.x * gridDim.x)
    printf("%c", message[i]);
}


int main(int argc, const char* argv[]) {
  std::string message;
  if (argc == 1) {
    message = "Hello world!";
  } else {
    message = argv[1];
  }
  
  char * buffer;
  hipMalloc(& buffer, message.size());
  hipMemcpy(buffer, message.data(), message.size(), hipMemcpyDefault);
  
  print<<<16,1>>>(buffer, message.size());
  hipDeviceSynchronize();
  hipGetLastError();
  std::cout << std::endl;
  
  print<<<4,4>>>(buffer, message.size());
  hipDeviceSynchronize();
  hipGetLastError();
  std::cout << std::endl;
  
  print<<<1,16>>>(buffer, message.size());
  hipDeviceSynchronize();
  hipGetLastError();
  std::cout << std::endl;
  
  hipFree(buffer);
}


